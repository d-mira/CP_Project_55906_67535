#include "hip/hip_runtime.h"
#include "histogram_eq_par.h"
#include <cub/cu.cuh>
#include <wb.h>

namespace cp::cub {
    constexpr auto HISTOGRAM_LENGTH = 256;

    static float inline prob(const int x, const int size) {
        return (float) x / (float) size;
    }

    __global__ void convert_to_uchar(const float* input_image_data, unsigned char* uchar_image, int size_channels) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size_channels) {
            uchar_image[idx] = static_cast<unsigned char>(255 * input_image_data[idx]);
        }
    }
    __global__ void compute_gray_image(const unsigned char* uchar_image, unsigned char* gray_image, int width, int height) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < width * height) {
            int r = uchar_image[3 * idx];
            int g = uchar_image[3 * idx + 1];
            int b = uchar_image[3 * idx + 2];
            gray_image[idx] = static_cast<unsigned char>(0.21 * r + 0.71 * g + 0.07 * b);
        }
    }
    __global__ void uchar_to_float(const unsigned char* uchar_image, float* output_image_data, int size_channels) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size_channels) {
            output_image_data[idx] = static_cast<float>(uchar_image[idx]) / 255.0f;
        }
    }

    __global__ void build_histogram(int* histogram, unsigned char* gray_image) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        std::fill(histogram, histogram+HISTOGRAM_LENGTH, 0);

        //Race conditions might apply, so maybe need to take care of that here

        if(idx < HISTOGRAM_LENGTH) {
            histogram[gray_image[idx]]++;
        }
    }

    //Pre-calculate an array of prob values, to improve runtime of cdf calculation
    __global__ void calc_prob_array() {

    }

    //Function calculated iteratively, by a single GPU thread
    __global__ void cdf_calculation(int* histogram, int size, float* cdf, float* prob) {
        cdf[0] = prob[0];
        for (int i = 1; i < HISTOGRAM_LENGTH; i++)
            cdf[i] = cdf[i - 1] + prob[0];
    }

    __global__ void min_cdf(float* cdf, float* cdf_min) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockIdx.x * gridDim.x;

        int numThreads = gridDim.x/blockDim.x;
        float *localMins = new float[numThreads];

        //Each thread will calculate a min value of the values it
        // got attributed to ("localMin")
        for(int i = idx; i < HISTOGRAM_LENGTH; i += stride) {
            localMins[idx] = std::min(localMins[idx], cdf[i]);
        }

        __syncThreads();

        //Calculate the total min
        for(int i = 0; i < numThreads; i++) {
            cdf_min = std::min(cdf_min, localMins[i]);
        }

    }

    //TODO - adicionar restantes cuda kernals histogram and cdf calculation e completar histogram_equalization

    //Should this be "__global__"? if so, do we need to extend the visibility
    // of its arguments to all threads or is this default behaviour?
      void histogram_equalization(const int width, const int height,
                                               const float *input_image_data,
                                               float *output_image_data,
                                               const std::shared_ptr<unsigned char[]> &uchar_image,
                                               const std::shared_ptr<unsigned char[]> &gray_image,
                                               int (&histogram)[HISTOGRAM_LENGTH],
                                               float (&cdf)[HISTOGRAM_LENGTH]) {
      //TODO
      }

      wbImage_t iterative_histogram_equalization_cub(wbImage_t &input_image, int iterations){
          const int width = wbImage_getWidth(input_image);
          const int height = wbImage_getHeight(input_image);
          const int size = width * height;
          const int size_channels = size * 3;

          wbImage_t output_image = wbImage_new(width, height, 3);
          float* input_image_data = wbImage_getData(input_image);
          float* output_image_data = wbImage_getData(output_image);

          for (int i = 0; i < iterations; i++) {
              histogram_equalization_par(width, height, input_image_data, output_image_data);
              input_image_data = output_image_data;
          }

          return output_image;
      }
  } 
     
  
} 
